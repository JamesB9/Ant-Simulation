#include "Colony.cuh"


Colony* createColoniesArray(int colonyCount) {
	Colony* colonies;
	hipMallocManaged(&colonies, colonyCount * sizeof(Colony));
	return colonies;
}
