#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// Title:            Ant Simulation
// Authors:           James Sergeant (100301636), James Burling (100266919),
//					  CallumGrimble (100243142) and Oliver Boys (100277126)
// File: EntitySystem.cu
// Description: The system that miniuplates the entities data.
//
// Change Log:
//	- 15/11/2021:JS - Added in block comments.
//
// Online sources:
//	- (URL)
//
//
//////////////////////////// 80 columns wide //////////////////////////////////
#include "EntitySystem.cuh"

MoveComponent* createMoveComponentArray(int n) {
	MoveComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(MoveComponent));
	return nArray;
}

SniffComponent* createSniffComponentArray(int n) {
	SniffComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(SniffComponent));
	return nArray;
}

ActivityComponent* createActivityComponentArray(int n) {
	ActivityComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(ActivityComponent));
	return nArray;
}

CollisionComponent* createCollisionComponentArray(int n) {
	CollisionComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(CollisionComponent));
	return nArray;
}

__device__ void move(MoveComponent& move, float deltaTime) {
	//Get random vector where {+1 < x > -1, +1 < y > -1}
	Vec2f randomDirection = randomInsideUnitCircle(&move.state);

	//Add randomDirection to the current direction
	move.direction = (move.direction + randomDirection * move.roamStrength);

	//Calculate speed based on direction
	Vec2f targetVelocity = move.direction * move.maxSpeed;

	//Calculate vector to turn to new direction
	Vec2f targetTurningForce = (targetVelocity - move.velocity) * move.turningForce;

	//Clamp new acceleration by maximum turning force
	Vec2f acceleration = clamp(targetTurningForce, move.turningForce);

	//Clamp new velocity to max speed
	move.velocity = clamp(move.velocity + acceleration * deltaTime, move.maxSpeed);
	move.position = move.position + (move.velocity * deltaTime);

	//Store current angle
	move.angle = atan2f(move.velocity.y, move.velocity.x);

	//Debug Output
	//printf("randx %f, randy %f \n", randomDirection.x, randomDirection.y);
	//printf("dirx %.2f, diry %.2f \n", move.direction.x, move.direction.y);
	//printf("tvx %.2f tvy %.2f ttfx %.2f ttfy %.2f \n", targetVelocity.x, targetVelocity.y, targetTurningForce.x, targetTurningForce.y);
	//printf("acx %.2f acy %.2f \n", acceleration.x, acceleration.y);
}

__device__ Vec2f getCellCoordinate(ItemGrid* itemGrid, float x, float y) {
	return { floorf(x / itemGrid->cellWidth), floorf(y / itemGrid->cellHeight) };
}

__device__ int getCellIndexDevice(ItemGrid* itemGrid, float x, float y) {
	return (floorf(y / itemGrid->cellHeight) * itemGrid->sizeX) + floorf(x / itemGrid->cellWidth);
}

__device__ Cell* getCellDevice(ItemGrid* itemGrid, float x, float y) {
	return &itemGrid->worldCells[getCellIndexDevice(itemGrid, x, y)];
}

__device__ int getCellIndex(ItemGrid* itemGrid, int x, int y) {
	return (floorf(y) * itemGrid->sizeX) + floorf(x);
}


__device__ void releasePheromone(ItemGrid* itemGrid, MoveComponent& move, ActivityComponent& activity, float deltaTime) {
	activity.timeSinceDrop += deltaTime;

	if (activity.timeSinceDrop > activity.timePerDrop && activity.dropStrength > 0.0f) {
		Cell* cell = getCellDevice(itemGrid, move.position.x, move.position.y);
		cell->pheromones[activity.currentActivity] += activity.dropStrength;
		activity.timeSinceDrop = 0;
	}
	activity.dropStrength -= activity.dropStrengthReduction * deltaTime;
}


__device__ float getPheromoneIntensitySample(ItemGrid* itemGrid, Vec2f position, int sampleRadius, int pheromoneType) {
	Vec2f cellCoordinate = getCellCoordinate(itemGrid, position.x, position.y);
	float totalIntensity = 0;
	for (int dx = cellCoordinate.x - sampleRadius; dx < cellCoordinate.x + sampleRadius; dx++) {
		for (int dy = cellCoordinate.y - sampleRadius; dy < cellCoordinate.y + sampleRadius; dy++) {
			totalIntensity += itemGrid->worldCells[getCellIndex(itemGrid, dx, dy)].pheromones[pheromoneType];
			if (pheromoneType == 1) {
				totalIntensity += itemGrid->worldCells[getCellIndex(itemGrid, dx, dy)].pheromones[2];//Food
			}
		}
	}

	return totalIntensity;
}

__device__ void senseHome(ItemGrid* itemGrid, Colony* colonies, MoveComponent& move, SniffComponent& sniff, ActivityComponent& activity, float deltaTime) {
	Vec2f home = { colonies[activity.colonyId].nestPositionX, colonies[activity.colonyId].nestPositionY };
	float distanceFromHome = getDistance(move.position, home);
	Vec2f vectorToHome;
	vectorToHome = home - move.position;
	vectorToHome = clamp(vectorToHome, 1.0f);

	float distanceFromRememberedFood = getDistance(move.position, activity.lastFoodPickup);
	Vec2f vectorToRememberedFood;
	vectorToRememberedFood = activity.lastFoodPickup - move.position;
	vectorToRememberedFood = clamp(vectorToRememberedFood, 1.0f);

	if (activity.currentActivity == 1) { move.direction = move.direction + (vectorToHome / (distanceFromHome/400.0f)); };
	if (activity.currentActivity == 0 && activity.lastFoodPickup.x != 0.0f && activity.lastFoodPickup.y != 0.0f) { move.direction = move.direction + (vectorToRememberedFood / (distanceFromRememberedFood / 400.0f)); };
}

__device__ void sniff(ItemGrid* itemGrid, Colony* colonies, MoveComponent& move, SniffComponent& sniff, ActivityComponent& activity, float deltaTime) {
	float distance = 15.0f;
	int sampleRadius = 5;
	// Get CELLS
	Cell* currentCell = getCellDevice(itemGrid, move.position.x, move.position.y);
	float baseAngle = atan2f(move.velocity.y, move.velocity.x) + (-90.0f * M_PI / 180);
	//printf("BA: %.2f\n", baseAngle);
	Vec2f leftVector;
	leftVector.x = (-distance * sin(baseAngle - M_PI_4));
	leftVector.y = (distance * cos(baseAngle - M_PI_4));
	
	float leftIntensity = getPheromoneIntensitySample(itemGrid, 
		leftVector + move.position,
		sampleRadius, 
		sniff.sniffPheromone);

	Vec2f rightVector;
	rightVector.x = (-distance * sin(baseAngle + M_PI_4));
	rightVector.y = (distance * cos(baseAngle + M_PI_4));
	
	float rightIntensity = getPheromoneIntensitySample(itemGrid,
		rightVector + move.position,
		sampleRadius, 
		sniff.sniffPheromone);

	Vec2f straightVector;
	straightVector.x = -distance * sin(baseAngle);
	straightVector.y = distance * cos(baseAngle);
	
	float straightIntensity = getPheromoneIntensitySample(itemGrid,
		straightVector + move.position,
		sampleRadius,
		sniff.sniffPheromone);

	float total_intensity = (straightIntensity) + (leftIntensity) + (rightIntensity);
	straightVector = straightVector * (straightIntensity / total_intensity);
	rightVector = rightVector * (rightIntensity / total_intensity);
	leftVector = leftVector * (leftIntensity / total_intensity);
	
	if (total_intensity > 0.0f) {
		//printf("%.2f - %.2f - %.2f\n", (leftIntensity / total_intensity), (straightIntensity / total_intensity), (rightIntensity / total_intensity));
		Vec2f finalVector = rightVector + leftVector + straightVector;
		//printf("%.2f, %.2f\n", finalVector.x, finalVector.y);
		move.direction = finalVector;
	}
	
	/*if (straightIntensity > leftIntensity && straightIntensity > rightIntensity) {
		move.direction = straightVector - move.position;
		//printf("Going straight!\n");
	}
	else if (leftIntensity > rightIntensity) {
		move.direction = leftVector - move.position;
		//printf("Turning left!\n");
	}
	else if (rightIntensity > leftIntensity) {
		move.direction = rightVector - move.position;
		//printf("Turning right!\n");
	}*/

	//printf("IL, IR: %.2f, %.2f\n", leftIntensity, rightIntensity);

	if (activity.currentActivity == 0 && currentCell->foodCount > 0.0f) { // FOOD FOUND!!
		currentCell->foodCount -= 1;
		activity.currentActivity = 1;
		sniff.sniffPheromone = 0;
		activity.dropStrength = activity.maxDropStrength;
		move.direction = { -move.direction.x, -move.direction.y };
		move.velocity = { 0,0 };
		//move.direction = { -move.direction.x, -move.direction.y };
		activity.lastFoodPickup = { move.position.x, move.position.y };
	}
	else if (activity.currentActivity == 1 && currentCell->foodCount > 0.0f) {
		activity.dropStrength = activity.maxDropStrength;
		activity.lastFoodPickup = { move.position.x, move.position.y };
	}

	float nestX = colonies[activity.colonyId].nestPositionX;
	float nestY = colonies[activity.colonyId].nestPositionY;
	float nestRadius = colonies[activity.colonyId].nestRadius;

	if (move.position.x > nestX - nestRadius && move.position.x < nestX + nestRadius && 
		move.position.y > nestY - nestRadius && move.position.y < nestY + nestRadius) { // HOME FOUND!!
		if (activity.currentActivity == 1) {
			move.direction = { -move.direction.x, -move.direction.y };
			move.velocity = { 0,0 };
		}
		activity.currentActivity = 0;
		sniff.sniffPheromone = 1;
		activity.dropStrength = activity.maxDropStrength;
		colonies->totalFood += 1;
	}

}

__device__ void detectWall(MoveComponent& move, CollisionComponent& collision, ActivityComponent& activity, Map* map, float deltaTime) {
	//Notes for wall detection
	//Cast ray out from and until you hit a 1 in the map
	//if distance from wall to ant is small enough
	//	1. Get the angle between the ant and the wall (1) OR Screen Border
	//	2. invert that angle based on what side of the wall you are on, find the inverse point (mirrored position) of the ant
	//	3. push that new location to the move function for turning

	/*Vec2f topLeft = {0.0f, 0.0f},
		bottomLeft = { 0.0f, 800.0f },
		topRight = { 800.0f, 0.0f },
		bottomRight = { 800.0f, 800.0f };
	Boundary lboundary = { topLeft, bottomLeft, 1 };
	Boundary rboundary = { bottomRight, topRight, 2 };
	Boundary tboundary = { topRight, topLeft, 3 };
	Boundary bboundary = { bottomLeft, bottomRight, 4 };
	Boundary boundaries[4] = { lboundary , rboundary , tboundary , bboundary };
	*/
	//ray position - move.position
	//ray's angle
	Vec2f angle = { cos(move.angle),  sin(move.angle) };
	float targetDistance = 1000000;
	int wallIndex = -1;
	//printf("at angle: %.2f, %.2f\n", angle.x, angle.y);

	for (int i = 0; i < map->wallCount; i++) {
		Boundary& wall = map->walls[i];
		const float x1 = wall.p1.x;
		const float y1 = wall.p1.y;
		const float x2 = wall.p2.x;
		const float y2 = wall.p2.y;

		const float x3 = move.position.x;
		const float y3 = move.position.y;
		const float x4 = move.position.x + (angle.x * 1000.0f);
		const float y4 = move.position.y + (angle.y * 1000.0f);

		const float den = (x1 - x2) * (y3 - y4) - (y1 - y2) * (x3 - x4);
		if (den == 0) { continue; }
		const float t = ((x1 - x3) * (y3 - y4) - (y1 - y3) * (x3 - x4)) / den;
		const float u = -((x1 - x2) * (y1 - y3) - (y1 - y2) * (x1 - x3)) / den;

		if (t > 0 && t < 1 && u > 0) {
			Vec2f targetPosition = { x1 + t * (x2 - x1) , y1 + t * (y2 - y1) };
			float distance = sqrtf(powf(targetPosition.x - move.position.x, 2.0f) + powf(targetPosition.y - move.position.y, 2.0f));

			if (distance < targetDistance) {//Calculate inverse angle
				wallIndex = i;
				targetDistance = distance;
				collision.targetPosition = targetPosition;
			}
		}
	}
	if (wallIndex != -1 && targetDistance < collision.collisionDistance) {
		//Calculate reflected angle
		Vec2f n = clamp(normaliseSurface(map->walls[wallIndex].p1, map->walls[wallIndex].p2), 1.0f);
		Vec2f u = n * (move.velocity.dotProduct(n) / n.dotProduct(n));
		Vec2f w = move.velocity - u;
		//Set reflected angle
		collision.refractionPosition = collision.targetPosition + (clamp(u-w, 1.0f) * targetDistance);
		//Set direction
		if (targetDistance < collision.collisionDistance) {
			move.direction = (u - w);
			//activity.dropStrength *= 0.5f * deltaTime;
		}
	}
	else if (wallIndex == -1) {
		move.position = { 400.0f, 400.0f };
	}
	//else {
	//	collision.refractionPosition = collision.targetPosition;
	//}
}

__global__ void simulateEntities(
	Entities* entities,
	float deltaTime,
	ItemGrid* itemGrid,
	Map* map,
	Colony* colonies)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x; // Index of the current thread within its block
	int stride = blockDim.x * gridDim.x; // Number of threads in the block
	for (int i = index; i < entities->entityCount; i += stride) { // For Each entity for this thread
		move(entities->moves[i], deltaTime);
		releasePheromone(itemGrid, entities->moves[i],  entities->activities[i],  deltaTime);
		sniff(itemGrid, colonies, entities->moves[i], entities->sniffs[i], entities->activities[i], deltaTime);
		senseHome(itemGrid, colonies, entities->moves[i], entities->sniffs[i], entities->activities[i], deltaTime);
		detectWall(entities->moves[i], entities->collisions[i], entities->activities[i], map, deltaTime);
	}
}

int simulateEntitiesOnGPU(Entities* entities, ItemGrid* itemGrid, Map* map, Colony* colonies, float deltaTime) {
	// Time Per Drop
	//ActivityComponent::timeSinceDrop +=

	int blockSize = 256;
	int numBlocks = (entities->entityCount + blockSize - 1) / blockSize;

	simulateEntities << <numBlocks, blockSize >> > (
		entities,
		deltaTime,
		itemGrid,
		map,
		colonies);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	return 0;
}


__global__ void setupStates(Entities* entities) {
	int index = blockIdx.x * blockDim.x + threadIdx.x; // Index of the current thread within its block
	int stride = blockDim.x * gridDim.x; // Number of threads in the block
	for (int i = index; i < entities->entityCount; i += stride) { // For Each entity for this thread
		//--RNG--
		hiprand_init(clock(), index, 0, &entities->moves[i].state);
	}
}

void setupStatesOnGPU(Entities* entities) {

	int blockSize = 256;
	int numBlocks = (entities->entityCount + blockSize - 1) / blockSize;

	setupStates << <numBlocks, blockSize >> > (entities);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
}

Entities* initEntities(Colony* colonies, int entityCount) {
	Entities* entities;
	hipMallocManaged(&entities, sizeof(Entities));
	entities->entityCount = entityCount;

	entities->moves = createMoveComponentArray(entities->entityCount);
	entities->sniffs = createSniffComponentArray(entities->entityCount);
	entities->activities = createActivityComponentArray(entities->entityCount);
	entities->collisions = createCollisionComponentArray(entities->entityCount);

	for (unsigned int i = 0; i < entities->entityCount; i++) {
		
		entities->activities[i].colonyId = 0; // CHANGE LATER

		entities->sniffs[i].sniffMaxDistance = Config::ANT_MAX_SNIFF_DISTANCE;
		entities->sniffs[i].sniffPheromone = FOUND_FOOD;

		entities->moves[i].position = { 
			colonies[entities->activities[i].colonyId].nestPositionX, 
			colonies[entities->activities[i].colonyId].nestPositionY 
		};
		entities->moves[i].direction = { 0.0f, 0.0f };
		entities->moves[i].velocity = { 0.0f, 0.0f };
		entities->moves[i].maxSpeed = Config::ANT_MAX_SPEED;
		entities->moves[i].turningForce = Config::ANT_TURN_FORCE;
		entities->moves[i].roamStrength = Config::ANT_ROAM_STRENGTH;

		entities->collisions[i].avoid = false;
		entities->collisions[i].targetPosition = {0.0f, 0.0f};
		entities->collisions[i].refractionPosition = { 0.0f, 0.0f };
		entities->collisions[i].collisionDistance = Config::ANT_COLLISION_DISTANCE;

		entities->activities[i].currentActivity = LEAVING_HOME;
		entities->activities[i].dropStrength = Config::INITIAL_DROP_STRENGTH;
		entities->activities[i].dropStrengthReduction = Config::DROP_STRENGTH_REDUCTION;
		entities->activities[i].timeSinceDrop = 0.0f;
		entities->activities[i].timePerDrop = Config::PHEROMONE_DROP_TIME;
		entities->activities[i].maxDropStrength = Config::INITIAL_DROP_STRENGTH;
		entities->activities[i].lastFoodPickup = { 0.0f, 0.0f };
	}

	return entities;
}
/*
int main() {
	Entities entities;
	initEntities(entities);
	printf("%f\n", entities.positions[0].x);
	simulateEntities(entities);
	printf("%f\n", entities.positions[0].x);
	return 0;
}*/
