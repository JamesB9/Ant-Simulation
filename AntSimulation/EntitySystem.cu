#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// Title:            Ant Simulation
// Authors:           James Sergeant (100301636), James Burling (100266919), 
//					  CallumGrimble (100243142) and Oliver Boys (100277126)
// File: EntitySystem.cu
// Description: The system that miniuplates the entities data.
// 
// Change Log:
//	- 15/11/2021:JS - Added in block comments.
//
// Online sources:  
//	- (URL)
// 
// 
//////////////////////////// 80 columns wide //////////////////////////////////
#include "EntitySystem.cuh"


MoveComponent* createMoveComponentArray(int n) {
	MoveComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(MoveComponent));
	return nArray;
}

SniffComponent* createSniffComponentArray(int n) {
	SniffComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(SniffComponent));
	return nArray;
}

ActivityComponent* createActivityComponentArray(int n) {
	ActivityComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(ActivityComponent));
	return nArray;
}

__device__ void move(MoveComponent& move, float deltaTime) {
	// Calculate Velocity
	float vx = deltaTime * move.speed * sin(move.rotation);
	float vy = deltaTime * move.speed * cos(move.rotation);
	move.x += vx;
	move.y += vy;
}

__device__ int getCellIndexDevice2(ItemGrid* itemGrid, float x, float y) {
	return (floorf(y) * itemGrid->worldX) + floorf(x);
}

__device__ Cell* getCellDevice2(ItemGrid* itemGrid, float x, float y) {
	return &itemGrid->worldCells[getCellIndexDevice2(itemGrid, x, y)];
}

__device__ void releasePheromone(ItemGrid* itemGrid, MoveComponent& move, ActivityComponent& activity) {
	Cell* cell = getCellDevice2(itemGrid, move.x, move.y);
	cell->pheromones[activity.currentActivity] += 0.5f;
}

__device__ void sniff(MoveComponent& move, SniffComponent& sniff, float deltaTime) {

}

__global__ void simulateEntities(
	MoveComponent* moves, 
	SniffComponent* sniffs,
	ActivityComponent* activities,
	int entityCount, 
	float deltaTime,
	ItemGrid* itemGrid) 
{

	int index = blockIdx.x * blockDim.x + threadIdx.x; // Index of the current thread within its block
	int stride = blockDim.x * gridDim.x; // Number of threads in the block
	for (int i = index; i < entityCount; i += stride) { // For Each entity for this thread
		move(moves[i], deltaTime);
		releasePheromone(itemGrid, moves[i], activities[i]);
	}
}

int simulateEntitiesOnGPU(Entities& entities, ItemGrid* itemGrid, float deltaTime) {
	// Run kernel on 1M elements on the CPU
	int blockSize = 256;
	int numBlocks = (entities.entityCount + blockSize - 1) / blockSize;
	
	simulateEntities << <numBlocks, blockSize >> > (
		entities.moves, 
		entities.sniffs, 
		entities.activities,
		entities.entityCount, 
		deltaTime,
		itemGrid);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	return 0;
}

int initEntities(Entities& entities) {
	entities.moves = createMoveComponentArray(entities.entityCount);
	entities.sniffs = createSniffComponentArray(entities.entityCount);
	entities.activities = createActivityComponentArray(entities.entityCount);

	for (unsigned int i = 0; i < entities.entityCount; i++) {
		entities.moves[i].x = 400;
		entities.moves[i].y = 400;
		entities.moves[i].rotation = 2 * M_PI * i / entities.entityCount;
		entities.moves[i].speed = 50;

		entities.sniffs[i].sniffMaxDistance = 5;
		entities.activities[i].currentActivity = LEAVING_HOME;
	}

	return 0;
}
/*
int main() {
	Entities entities;
	initEntities(entities);
	printf("%f\n", entities.positions[0].x);
	simulateEntities(entities);
	printf("%f\n", entities.positions[0].x);
	return 0;
}*/