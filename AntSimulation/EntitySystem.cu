#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// Title:            Ant Simulation
// Authors:           James Sergeant (100301636), James Burling (100266919),
//					  CallumGrimble (100243142) and Oliver Boys (100277126)
// File: EntitySystem.cu
// Description: The system that miniuplates the entities data.
//
// Change Log:
//	- 15/11/2021:JS - Added in block comments.
//
// Online sources:
//	- (URL)
//
//
//////////////////////////// 80 columns wide //////////////////////////////////
#include "EntitySystem.cuh"

MoveComponent* createMoveComponentArray(int n) {
	MoveComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(MoveComponent));
	return nArray;
}

SniffComponent* createSniffComponentArray(int n) {
	SniffComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(SniffComponent));
	return nArray;
}

ActivityComponent* createActivityComponentArray(int n) {
	ActivityComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(ActivityComponent));
	return nArray;
}

CollisionComponent* createCollisionComponentArray(int n) {
	CollisionComponent* nArray;
	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged(&nArray, n * sizeof(CollisionComponent));
	return nArray;
}

__device__ void move(MoveComponent& move, hiprandState* state, float deltaTime) {
	//Get random vector where {+1 < x > -1, +1 < y > -1}
	Vec2f randomDirection = randomInsideUnitCircle(state);

	//Add randomDirection to the current direction
	move.direction = (move.direction + randomDirection * move.roamStrength);

	//Calculate speed based on direction
	Vec2f targetVelocity = move.direction * move.maxSpeed;

	//Calculate vector to turn to new direction
	Vec2f targetTurningForce = (targetVelocity - move.velocity) * move.turningForce;

	//Clamp new acceleration by maximum turning force
	Vec2f acceleration = clamp(targetTurningForce, move.turningForce);

	//Store current angle
	move.angle = atan2f(move.velocity.y, move.velocity.x);

	//Clamp new velocity to max speed
	move.velocity = clamp(move.velocity + acceleration * deltaTime, move.maxSpeed);
	move.position = move.position + (move.velocity * deltaTime);

	//Debug Output
	//printf("randx %f, randy %f \n", randomDirection.x, randomDirection.y);
	//printf("dirx %.2f, diry %.2f \n", move.direction.x, move.direction.y);
	//printf("tvx %.2f tvy %.2f ttfx %.2f ttfy %.2f \n", targetVelocity.x, targetVelocity.y, targetTurningForce.x, targetTurningForce.y);
	//printf("acx %.2f acy %.2f \n", acceleration.x, acceleration.y);
}

__device__ Vec2f getCellCoordinate(ItemGrid* itemGrid, float x, float y) {
	return { floorf(x / itemGrid->cellWidth), floorf(y / itemGrid->cellHeight) };
}

__device__ int getCellIndexDevice(ItemGrid* itemGrid, float x, float y) {
	return (floorf(y / itemGrid->cellHeight) * itemGrid->sizeX) + floorf(x / itemGrid->cellWidth);
}

__device__ Cell* getCellDevice(ItemGrid* itemGrid, float x, float y) {
	return &itemGrid->worldCells[getCellIndexDevice(itemGrid, x, y)];
}

__device__ int getCellIndex(ItemGrid* itemGrid, int x, int y) {
	return (floorf(y) * itemGrid->sizeX) + floorf(x);
}


__device__ void releasePheromone(ItemGrid* itemGrid, MoveComponent& move, ActivityComponent& activity, float deltaTime) {
	activity.timeSinceDrop += deltaTime;

	if (activity.timeSinceDrop > activity.timePerDrop && activity.dropStrength > 0.0f) {
		Cell* cell = getCellDevice(itemGrid, move.position.x, move.position.y);
		if (activity.currentActivity == 0) { // IF LEAVING HOME
			if (cell->pheromones[0] >= cell->pheromones[1]) { // Dont drop home pheromone onto food path
				cell->pheromones[activity.currentActivity] += activity.dropStrength;
				activity.timeSinceDrop = 0;
			}
		}
	}
	activity.dropStrength -= activity.dropStrengthReduction * deltaTime;
}


__device__ float getPheromoneIntensitySample(ItemGrid* itemGrid, float centerX, float centerY, int sampleRadius, int pheromoneType) {
	Vec2f cellCoordinate = getCellCoordinate(itemGrid, centerX, centerY);
	float totalIntensity = 0;
	for (int dx = cellCoordinate.x - sampleRadius; dx < cellCoordinate.x + sampleRadius; dx++) {
		for (int dy = cellCoordinate.y - sampleRadius; dy < cellCoordinate.y + sampleRadius; dy++) {
			totalIntensity += itemGrid->worldCells[getCellIndex(itemGrid, dx, dy)].pheromones[pheromoneType];
		}
	}

	return totalIntensity;
}

__device__ void sniff(ItemGrid* itemGrid, Colony* colonies, MoveComponent& move, SniffComponent& sniff, ActivityComponent& activity, float deltaTime) {
	float distance = 15;
	int sampleRadius = 3;
	// Get CELLS
	Cell* currentCell = getCellDevice(itemGrid, move.position.x, move.position.y);

	float angle = move.angle - M_PI_2;
	float leftIntensity = getPheromoneIntensitySample(itemGrid, 
		move.position.x + distance * cosf(angle), 
		move.position.y + distance * sinf(angle), 
		sampleRadius, sniff.sniffPheromone);

	angle = move.angle + M_PI_2;
	float rightIntensity = getPheromoneIntensitySample(itemGrid,
		move.position.x + distance * cosf(angle),
		move.position.y + distance * sinf(angle),
		sampleRadius, sniff.sniffPheromone);
	//printf("%f, %f\n", leftIntensity, rightIntensity);
	angle = move.angle;
	/*
	float middleIntensity = getPheromoneIntensitySample(itemGrid,
		move.position.x + distance * sin(angle),
		move.position.y + distance * cos(angle),
		sampleRadius, sniff.sniffPheromone);
	*/

	if (leftIntensity > rightIntensity) {
		angle = move.angle - M_PI_2;
		move.direction = { cosf(angle) , sinf(angle) };
	}
	else if (rightIntensity > leftIntensity) {
		angle = move.angle + M_PI_2;
		move.direction = { cosf(angle) , sinf(angle) };
	}

	if (activity.currentActivity == 0 && currentCell->foodCount > 0.0f) { // FOOD FOUND!!
		currentCell->foodCount -= 1;
		activity.currentActivity = 1;
		sniff.sniffPheromone = 0;
		activity.dropStrength = activity.maxDropStrength;
		move.direction = { -move.direction.x, -move.direction.y };
	}

	float nestX = colonies[activity.colonyId].nestPositionX;
	float nestY = colonies[activity.colonyId].nestPositionY;
	float nestRadius = colonies[activity.colonyId].nestRadius;

	if (move.position.x > nestX - nestRadius && move.position.x < nestX + nestRadius && 
		move.position.y > nestY - nestRadius && move.position.y < nestY + nestRadius) { // HOME FOUND!!
		activity.currentActivity = 0;
		sniff.sniffPheromone = 1;
		activity.dropStrength = activity.maxDropStrength;
		colonies->totalFood += 1;
	}
}

__device__ void detectWall(MoveComponent& move, CollisionComponent& collision, ActivityComponent& activity, Map* map, float deltaTime) {
	//Notes for wall detection
	//Cast ray out from and until you hit a 1 in the map
	//if distance from wall to ant is small enough
	//	1. Get the angle between the ant and the wall (1) OR Screen Border
	//	2. invert that angle based on what side of the wall you are on, find the inverse point (mirrored position) of the ant
	//	3. push that new location to the move function for turning

	/*Vec2f topLeft = {0.0f, 0.0f},
		bottomLeft = { 0.0f, 800.0f },
		topRight = { 800.0f, 0.0f },
		bottomRight = { 800.0f, 800.0f };
	Boundary lboundary = { topLeft, bottomLeft, 1 };
	Boundary rboundary = { bottomRight, topRight, 2 };
	Boundary tboundary = { topRight, topLeft, 3 };
	Boundary bboundary = { bottomLeft, bottomRight, 4 };
	Boundary boundaries[4] = { lboundary , rboundary , tboundary , bboundary };
	*/
	//ray position - move.position
	//ray's angle
	Vec2f angle = { cos(move.angle),  sin(move.angle) };
	float targetDistance = 1000000;
	int wallIndex = -1;
	//printf("at angle: %.2f, %.2f\n", angle.x, angle.y);

	for (int i = 0; i < map->wallCount; i++) {
		Boundary& wall = map->walls[i];
		const float x1 = wall.p1.x;
		const float y1 = wall.p1.y;
		const float x2 = wall.p2.x;
		const float y2 = wall.p2.y;

		const float x3 = move.position.x;
		const float y3 = move.position.y;
		const float x4 = move.position.x + (angle.x * 1000.0f);
		const float y4 = move.position.y + (angle.y * 1000.0f);

		const float den = (x1 - x2) * (y3 - y4) - (y1 - y2) * (x3 - x4);
		if (den == 0) { continue; }
		const float t = ((x1 - x3) * (y3 - y4) - (y1 - y3) * (x3 - x4)) / den;
		const float u = -((x1 - x2) * (y1 - y3) - (y1 - y2) * (x1 - x3)) / den;

		if (t > 0 && t < 1 && u > 0) {
			Vec2f targetPosition = { x1 + t * (x2 - x1) , y1 + t * (y2 - y1) };
			float distance = sqrtf(powf(targetPosition.x - move.position.x, 2.0f) + powf(targetPosition.y - move.position.y, 2.0f));

			if (distance < targetDistance) {//Calculate inverse angle
				wallIndex = i;
				targetDistance = distance;
				collision.targetPosition = targetPosition;
			}
		}
	}
	if (wallIndex != -1 && targetDistance < collision.collisionDistance) {
		//Calculate reflected angle
		Vec2f n = clamp(normaliseSurface(map->walls[wallIndex].p1, map->walls[wallIndex].p2), 1.0f);
		Vec2f u = n * (move.velocity.dotProduct(n) / n.dotProduct(n));
		Vec2f w = move.velocity - u;
		//Set reflected angle
		collision.refractionPosition = collision.targetPosition + (clamp(u-w, 1.0f) * targetDistance);
		//Set direction
		if (targetDistance < collision.collisionDistance) {
			move.direction = (u - w);
			//activity.dropStrength *= 0.5f * deltaTime;
		}
	}
	else if (wallIndex == -1) {
		move.position = { 400.0f, 400.0f };
	}
	//else {
	//	collision.refractionPosition = collision.targetPosition;
	//}
}

__global__ void simulateEntities(
	Entities* entities,
	float deltaTime,
	ItemGrid* itemGrid,
	Map* map,
	Colony* colonies)
{
	//--RNG--
	hiprandState state;
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(clock(), id, 0, &state);

	int index = blockIdx.x * blockDim.x + threadIdx.x; // Index of the current thread within its block
	int stride = blockDim.x * gridDim.x; // Number of threads in the block
	for (int i = index; i < entities->entityCount; i += stride) { // For Each entity for this thread
		move(entities->moves[i], &state ,deltaTime);
		releasePheromone(itemGrid, entities->moves[i],  entities->activities[i],  deltaTime);
		sniff(itemGrid, colonies, entities->moves[i], entities->sniffs[i], entities->activities[i], deltaTime);
		detectWall(entities->moves[i], entities->collisions[i], entities->activities[i], map, deltaTime);
	}
}

int simulateEntitiesOnGPU(Entities* entities, ItemGrid* itemGrid, Map* map, Colony* colonies, float deltaTime) {
	// Time Per Drop
	//ActivityComponent::timeSinceDrop +=

	int blockSize = 256;
	int numBlocks = (entities->entityCount + blockSize - 1) / blockSize;

	simulateEntities << <numBlocks, blockSize >> > (
		entities,
		deltaTime,
		itemGrid,
		map,
		colonies);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	//std::cout << entities.moves[0].position.x << ", " << entities.moves[0].position.y << std::endl;

	return 0;
}

Entities* initEntities(Colony* colonies, int entityCount) {
	Entities* entities;
	hipMallocManaged(&entities, sizeof(Entities));
	entities->entityCount = entityCount;

	entities->moves = createMoveComponentArray(entities->entityCount);
	entities->sniffs = createSniffComponentArray(entities->entityCount);
	entities->activities = createActivityComponentArray(entities->entityCount);
	entities->collisions = createCollisionComponentArray(entities->entityCount);

	for (unsigned int i = 0; i < entities->entityCount; i++) {
		
		entities->activities[i].colonyId = 0; // CHANGE LATER

		entities->sniffs[i].sniffMaxDistance = Config::ANT_MAX_SNIFF_DISTANCE;
		entities->sniffs[i].sniffPheromone = FOUND_FOOD;

		entities->moves[i].position = { 
			colonies[entities->activities[i].colonyId].nestPositionX, 
			colonies[entities->activities[i].colonyId].nestPositionY 
		};
		entities->moves[i].direction = { 0.0f, 0.0f };
		entities->moves[i].velocity = { 0.0f, 0.0f };
		entities->moves[i].maxSpeed = Config::ANT_MAX_SPEED;
		entities->moves[i].turningForce = Config::ANT_TURN_FORCE;
		entities->moves[i].roamStrength = Config::ANT_ROAM_STRENGTH;

		entities->collisions[i].avoid = false;
		entities->collisions[i].targetPosition = {0.0f, 0.0f};
		entities->collisions[i].refractionPosition = { 0.0f, 0.0f };
		entities->collisions[i].collisionDistance = Config::ANT_COLLISION_DISTANCE;

		entities->activities[i].currentActivity = LEAVING_HOME;
		entities->activities[i].dropStrength = Config::INITIAL_DROP_STRENGTH;
		entities->activities[i].dropStrengthReduction = Config::DROP_STRENGTH_REDUCTION;
		entities->activities[i].timeSinceDrop = 0.0f;
		entities->activities[i].timePerDrop = Config::PHEROMONE_DROP_TIME;
		entities->activities[i].maxDropStrength = Config::INITIAL_DROP_STRENGTH;
	}

	return entities;
}
/*
int main() {
	Entities entities;
	initEntities(entities);
	printf("%f\n", entities.positions[0].x);
	simulateEntities(entities);
	printf("%f\n", entities.positions[0].x);
	return 0;
}*/
