#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// Title:            Ant Simulation
// Authors:           James Sergeant (100301636), James Burling (100266919), 
//					  CallumGrimble (100243142) and Oliver Boys (100277126)
// File: Utilities.cu
// Description: The item grid implmentation for the simulation.
// 
// Change Log:
//	- 15/11/2021:JS - Added in block comments.
//
// Online sources:  
//	- (URL)
// 
// 
//////////////////////////// 80 columns wide //////////////////////////////////
#include "itemgrid.cuh"

Cell* createItemGridCellArray(int worldSize) {
	Cell* nArray;
	hipMallocManaged(&nArray, worldSize * sizeof(Cell));
	return nArray;
}

ItemGrid* initItemGrid(int worldX, int worldY) {
	ItemGrid* itemGrid;
	hipMallocManaged(&itemGrid, sizeof(ItemGrid));

	itemGrid->worldX = worldX;
	itemGrid->worldY = worldY;
	itemGrid->totalCells = worldX * worldY;

	itemGrid->worldCells = createItemGridCellArray(itemGrid->totalCells);
	for (int i = 0; i < itemGrid->totalCells; i++) {
		itemGrid->worldCells[i].foodCount = 0.0f;
		itemGrid->worldCells[i].pheromones[0] = 0.0f;
		itemGrid->worldCells[i].pheromones[1] = 0.0f;

		itemGrid->worldCells[i].timePerDrop = 3.0f;
		itemGrid->worldCells[i].timeSinceDrop = 0.0f;
	}
	return itemGrid;
}

Cell* getCell(ItemGrid& itemGrid, float x, float y) {
	//Take X and Y, convert to 2D reference
	//int posx = floorf(x);
	//int posy = floorf(y);
	//int index = posy * itemGrid.worldX;
	//index += posx;


	return &itemGrid.worldCells[getCellIndex(itemGrid, x, y)];
}

int getCellIndex(ItemGrid& itemGrid, float x, float y) {
	return (floorf(y) * itemGrid.worldX) + floorf(x);
}

void updateCell(Cell& cell, float deltaTime) {
	if (cell.pheromones[0] > 0.0f || cell.pheromones[1] > 0.0f) {
		cell.timeSinceDrop += deltaTime;
		if (cell.timeSinceDrop > cell.timePerDrop) {
			cell.pheromones[0] > 0.1f ? cell.pheromones[0] -= 0.1f : cell.pheromones[0] = 0.0f;
			cell.pheromones[1] > 0.1f ? cell.pheromones[1] -= 0.1f : cell.pheromones[1] = 0.0f;

			cell.timeSinceDrop = 0.0f;
		}
	}
}


int getCellIndex(ItemGrid* itemGrid, float mapx, float mapy) {
	float widthOfCell = 800.0f / itemGrid->worldX;
	float heightOfCell = 800.0f / itemGrid->worldY;
	return (floorf(mapy / heightOfCell) * itemGrid->worldX) + floorf(mapx / widthOfCell);
}

Cell* getCell(ItemGrid* itemGrid, float mapx, float mapy) {
	return &itemGrid->worldCells[getCellIndex(itemGrid, mapx, mapy)];
}