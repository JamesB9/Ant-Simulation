#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// Title:            Ant Simulation
// Authors:           James Sergeant (100301636), James Burling (100266919), 
//					  CallumGrimble (100243142) and Oliver Boys (100277126)
// File: Utilities.cu
// Description: The item grid implmentation for the simulation.
// 
// Change Log:
//	- 15/11/2021:JS - Added in block comments.
//
// Online sources:  
//	- (URL)
// 
// 
//////////////////////////// 80 columns wide //////////////////////////////////
#include "itemgrid.cuh"

Cell* createItemGridCellArray(int worldSize) {
	Cell* nArray;
	hipMallocManaged(&nArray, worldSize * sizeof(Cell));
	return nArray;
}

int initItemGrid(ItemGrid& itemGrid, int worldX, int worldY) {
	itemGrid.worldCells = createItemGridCellArray(worldX * worldY);
	itemGrid.worldX = worldX;
	itemGrid.worldY = worldY;
	itemGrid.totalCells = worldX * worldY;

	return 0;
}

Cell* getCell(ItemGrid& itemGrid, float x, float y) {
	//Take X and Y, convert to 2D reference
	int posx = floorf(x);
	int posy = floorf(y);
	int index = posy * itemGrid.worldX;
	index += posx;


	return &itemGrid.worldCells[index];
}